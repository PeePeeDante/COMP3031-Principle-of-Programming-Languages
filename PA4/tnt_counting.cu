#include "hip/hip_runtime.h"
#include <iostream>
#include "helpers.h"
using namespace std;

/* you can define data structures and helper functions here */

__device__ void c6ring_dfs(int vertex_id, int depth, int* dcc, int c_c_size, int visited[], 
						   int neighbors[], int num_neighbors, int** &results, int &num_results){

	visited[depth]=vertex_id;
	depth++;

	if (depth<6){

		for (int i=0; i<num_neighbors; i++){

			int next_vertex_id = neighbors[i];

			// Check if visited
			bool visit = 0;
			for (int j=0; j<6; j++){
				if (visited[j]==next_vertex_id){
					visit = 1;
					break;
				}
			}
			if (visit) continue;

			// Initialize neighbor info
			int num_next_neighbors = 0;
			for (int j=0; j<c_c_size; j++){
				if (dcc[j]==next_vertex_id) num_next_neighbors++;
			}

			//const int NUM_NEXT_NEIGHBORS = num_next_neighbors;
			//int next_neighbors[NUM_NEXT_NEIGHBORS];
			int next_neighbors[10];
			int p = 0;
			for (int j=0; j<c_c_size; j++){
				if (dcc[j]==next_vertex_id){
					next_neighbors[p] = dcc[j+c_c_size];
					p++;
				}
			}

			// Next dfs
			c6ring_dfs(next_vertex_id, depth, dcc, c_c_size, visited, 
					   next_neighbors, num_next_neighbors, results, num_results);
		}
	}

	// Check if c6ring is found
	else if (depth == 6) {

		bool found = 0;

		// Check if visited[0] is in neighbors
		for (int i=0; i<num_neighbors; i++){
			if (neighbors[i]==visited[0]){
				found = 1;
				break;
			}
		} 

		// Update result, found
		if (found == 1){

			int** temp = new int* [num_results+1];

			for (int i=0; i<num_results; i++){
				temp[i] = new int [6];
				for (int j = 0; j<6; j++){
					temp[i][j] = results[i][j];
				}
				delete[] results[i];
			}

			temp[num_results] = new int [6];
			for (int i=0; i<6; i++){
				temp[num_results][i] = visited[i]; 
			}

			if (results!=nullptr) delete[] results;
			
			results = temp;
			num_results++;
		}
	}

	// Backtrack
	visited[depth-1] = -1;

	// Depth is pbv no need update
}

__device__ void no2_search(int* dcn, int c_n_size, int* dno, int n_o_size, 
						   int** results, int num_results, int** &tnt_results, int& num_tnt_results){
	
	// Traverse each c6ring results
	for (int i=0; i<num_results; i++){

		// Store n vertex id if exists
		int cn[3] = {-1,-1,-1};

		// Find cn of c6ring[0], c6ring[2], c6ring[4]
		for (int j=0; j<3; j++){

			int c_vertex = results[i][2*j];
			for (int k=0; k<c_n_size; k++){

				if (c_vertex==dcn[k]){
					cn[j] = dcn[k+c_n_size];
					break;
				}
			}
		}

		// Check if all cn[3] not -1
		bool valid = 1;
		for (int j=0; j<3; j++){
			if (cn[j]==-1){
				valid = 0;
				break;
			}
		}

		// Check no2
		if (valid){

			// Find 2 os for each n in cn
			int no2[3][2] = {-1,-1,-1,-1,-1,-1};
			for (int j=0; j<3; j++){

				int l = 0;
				for (int k=0; k<n_o_size; k++){

					if (cn[j]==dno[k]){
						no2[j][l] = dno[k+n_o_size];
						l++;
					}
					if (l==2) break;
				}
			}

			// Check if no2 all not -1
			bool valid2 = 1;
			for (int j=0; j<3; j++){
				for (int k=0; k<2; k++){
					if (no2[j][k]==-1){
						valid2 = 0;
						break;
					}
				}
				if (valid2==0) break;
			}

			if (valid2){
				int** temp = new int* [8*(num_tnt_results+1)];

				for (int j=0; j<8*num_tnt_results; j++){
					temp[j] = new int [15];
					for (int k = 0; k<15; k++){
						temp[j][k] = tnt_results[j][k];
					}
					delete[] tnt_results[j];
				}

				for (int j=0; j<8; j++){
					temp[8*(num_tnt_results)+j] = new int [15];
					for (int k=0; k<6; k++) temp[j][k] = results[i][k];
					for (int k=0; k<3; k++) temp[j][k+6] = cn[k];
					for (int k=0; k<3; k++){
						for (int l=0; l<2; l++){
							temp[j][2*k+l+9] = no2[k][l];
						}
					}

					// For each cno2, write 2 different combination
					// ab ba ab ba ab ba ab ba
					// cd cd dc dc cd cd dc dc
					// ef ef ef ef fe fe fe fe 
					int temp1 = no2[0][0];
					no2[0][0] = no2[0][1];
					no2[0][1] = temp1;

					if (j%2==0){
						int temp2 = no2[1][0];
						no2[1][0] = no2[1][1];
						no2[1][1] = temp2;
					}

					if (j%4==0){
						int temp3 = no2[2][0];
						no2[2][0] = no2[2][1];
						no2[2][1] = temp3;
					}
				}

				if (tnt_results!=nullptr) delete[] tnt_results;

				tnt_results = temp;
				num_tnt_results = num_tnt_results + 8;
			}
		}
	}
}

__device__ void no2_count(int* dcn, int c_n_size, int* dno, int n_o_size, 
						  int** results, int num_results, int& num_tnt_results){
	
	// Traverse each c6ring results
	for (int i=0; i<num_results; i++){

		// Store n vertex id if exists
		int cn[3] = {-1,-1,-1};

		// Find cn of c6ring[0], c6ring[2], c6ring[4]
		for (int j=0; j<3; j++){

			int c_vertex = results[i][2*j];
			for (int k=0; k<c_n_size; k++){

				if (c_vertex==dcn[k]){
					cn[j] = dcn[k+c_n_size];
					break;
				}
			}
		}

		// Check if all cn[3] not -1
		bool valid = 1;
		for (int j=0; j<3; j++){
			if (cn[j]==-1){
				valid = 0;
				break;
			}
		}

		// Check no2
		if (valid){

			//printf("debug4: valid!\t");

			// Find 2 os for each n in cn
			int no2[3][2] = {-1,-1,-1,-1,-1,-1};
			for (int j=0; j<3; j++){

				int l = 0;
				for (int k=0; k<n_o_size; k++){

					if (cn[j]==dno[k]){
						no2[j][l] = dno[k+n_o_size];
						l++;
					}
					if (l==2) break;
				}
			}

			// Check if no2 all not -1
			bool valid2 = 1;
			for (int j=0; j<3; j++){
				for (int k=0; k<2; k++){
					if (no2[j][k]==-1){
						valid2 = 0;
						break;
					}
				}
				if (valid2==0) break;
			}

			if (valid2){
				num_tnt_results = num_tnt_results+8;
				//printf("debug5: %d\n", num_tnt_results);
			} 
		}
	}
}

__global__ void tnt_search(int* dcc, int* dcn, int* dno, int c_c_size, int c_n_size, int n_o_size, 
						   int* d_thread_results, int gridSize){
	
	//const int SHARED_DCC_SIZE = 2*c_c_size;
	//const int SHARED_DCN_SIZE = 2*c_n_size;
	//const int SHARED_DNO_SIZE = 2*n_o_size;
	//__shared__ int shared_dcc[SHARED_DCC_SIZE];
	//__shared__ int shared_dcn[SHARED_DCN_SIZE];
	//__shared__ int shared_dno[SHARED_DNO_SIZE];
	
	/*
	__shared__ int shared_dcc[16384];
	__shared__ int shared_dcn[16384];
	__shared__ int shared_dno[16384];

	for (int i=0; i<2*c_c_size; i++){
		shared_dcc[i] = dcc[i];
	}
	for (int i=0; i<2*c_n_size; i++){
		shared_dcn[i] = dcn[i];
	}
	for (int i=0; i<2*n_o_size; i++){
		shared_dno[i] = dno[i];
	}

	__syncthreads();
	*/

	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int i = 0;

	// Some threads have to take multiple tasks
	while (i*gridSize+tid < c_c_size){

		int idx = i*gridSize + tid;

		// Initialize visited array
		int visited[6] = {-1,-1,-1,-1,-1,-1};

		// Starting C only visits direct neighbor to avoid duplicates
		int cc_start = dcc[idx];
		int neighbors[1] = { dcc[idx+c_c_size] };

		// Call c6ring_dfs()
		int** results = nullptr;
		int num_results = 0;

		c6ring_dfs(cc_start, 0, dcc, c_c_size, visited, neighbors, 1, results, num_results);
		// Return results -> arrays of c6rings
		// Return num_results
		// printf("debug3: %d", num_results);

		// Call no2_search() on c6ring results
		int num_tnt_results = 0;

		if (results!=nullptr){
			no2_count(dcn, c_n_size, dno, n_o_size, results, num_results, num_tnt_results);
		}
		// Return num_tnt_results

		// printf("debug2: %d", num_tnt_results);
		d_thread_results[idx] = num_tnt_results;

		i++;
	}
}

__global__ void tnt_results(int* dcc, int* dcn, int* dno, int c_c_size, int c_n_size, int n_o_size, 
						    int* d_thread_results, int* d_thread_tnt_map, int thread_tnt_map_size, 
						    int* d_final_results, int final_result_size, int gridSize){

	//const int SHARED_DCC_SIZE = 2*c_c_size;
	//const int SHARED_DCN_SIZE = 2*c_n_size;
	//const int SHARED_DNO_SIZE = 2*n_o_size;
	//__shared__ int shared_dcc[SHARED_DCC_SIZE];
	//__shared__ int shared_dcn[SHARED_DCN_SIZE];
	//__shared__ int shared_dno[SHARED_DNO_SIZE];

	/*
	__shared__ int shared_dcc[16384];
	__shared__ int shared_dcn[];
	__shared__ int shared_dno[16384];

	for (int i=0; i<2*c_c_size; i++){
		shared_dcc[i] = dcc[i];
	}
	for (int i=0; i<2*c_n_size; i++){
		shared_dcn[i] = dcn[i];
	}
	for (int i=0; i<2*n_o_size; i++){
		shared_dno[i] = dno[i];
	}

	__syncthreads();
	*/

	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int i = 0;

	// Some threads have to take multiple tasks
	while (i*gridSize+tid < thread_tnt_map_size){

		int idx = i*gridSize + tid;

		// Find what C should this thread access
		int matching_c = d_thread_tnt_map[idx];

		// int preserve_space = d_thread_results[matching_c];
		int start = 0;
		for (int j=0; j<idx; j++){
			start = start + d_thread_results[d_thread_tnt_map[j]];
			// j=0~idx 				-> go through previous threads
			// d_thread_tnt_map[] 	-> find pos on c_c[] that has tnt
			// d_thread_results[]	-> find how many tnts for that C (~ preserved space)
		}
		// Want to write results between final_results[start : start+preserve_space]

		// Initialize visited array
		int visited[6] = {-1,-1,-1,-1,-1,-1};

		// Starting C only visits direct neighbor to avoid duplicates
		int cc_start = dcc[matching_c];
		int neighbors[1] = { dcc[matching_c+c_c_size] };

		// Call c6ring_dfs()
		int** results = nullptr;
		int num_results = 0;

		c6ring_dfs(cc_start, 0, dcc, c_c_size, visited, neighbors, 1, results, num_results);
		// Return results -> arrays of c6rings
		// Return num_results

		// printf("debug8: %d", num_results);

		// Call no2_search() on c6ring results
		int** tnt_results = nullptr;
		int num_tnt_results = 0;

		if (results!=nullptr){
			no2_search(dcn, c_n_size, dno, n_o_size, results, num_results, tnt_results, num_tnt_results);
		}
		// Return tnt_results -> arrays of tnt
		// Return num_tnt_results
		// printf("debug7: %d", num_tnt_results);

		for (int j=0; j<num_tnt_results; j++){
			for(int k=0; k<15; k++){
				d_final_results[start+j+k*final_result_size] = tnt_results[j][k];
			}
			//printf("debug \n");
		}

		i++;
	}

}

/**
 * please remember to set final_results and final_result_size 
 * before return.
 */
void tnt_counting(int num_blocks_per_grid, int num_threads_per_block,
        int* c_c, int* c_n, int* c_h, int* n_o,
        int c_c_size, int c_n_size, int c_h_size, int n_o_size,
        int* &final_results, int &final_result_size) {
    
    // Initialize
    int gridSize = num_blocks_per_grid * num_threads_per_block;
    int *dcc, *dcn, *dno;

	int *thread_results = new int[c_c_size];
	for (int i=0; i<c_c_size; i++) thread_results[i] = 0;

    int *d_thread_results;

    // Allocate vectors in device memory
    hipMalloc((void**) &dcc, c_c_size*2*sizeof(int));
    hipMalloc((void**) &dcn, c_n_size*2*sizeof(int));
    hipMalloc((void**) &dno, n_o_size*2*sizeof(int));
    hipMalloc((void**) &d_thread_results, c_c_size*sizeof(int));

    // Copy vectors from host to device global memory
    hipMemcpy(dcc, c_c, c_c_size*2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dcn, c_n, c_n_size*2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dno, n_o, n_o_size*2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_thread_results, thread_results, c_c_size*sizeof(int), hipMemcpyHostToDevice);

    // Step1: find final_result_size, return an array of count of result among all c_c
    tnt_search <<< num_blocks_per_grid, num_threads_per_block >>> (dcc, dcn, dno, c_c_size, c_n_size, n_o_size, d_thread_results, gridSize);

    // Copy vectors from device to host
    hipMemcpy(thread_results, d_thread_results, c_c_size*sizeof(int), hipMemcpyDeviceToHost);

    // Set final_result_size
    for (int i=0; i<c_c_size; i++){
    	final_result_size+=thread_results[i];
    }

	// cout << "debug1: " << final_result_size << endl;

    // Initialize 
    int thread_tnt_map_size = 0;
    for (int i=0; i<c_c_size; i++){
    	if (thread_results[i]) thread_tnt_map_size++;
    }
    int thread_tnt_map[thread_tnt_map_size];
    int p=0;
    for (int i=0; i<c_c_size; i++){
    	if (thread_results[i]){
    		thread_tnt_map[p] = i;
    		p++;
    	}
    }

    final_results = new int [final_result_size*15];
    int* d_final_results, *d_thread_tnt_map;

    // Allocate vectors in device memory
    hipMalloc((void**) &d_final_results, final_result_size*15*sizeof(int));
    hipMalloc((void**) &d_thread_tnt_map, thread_tnt_map_size*sizeof(int));

	/* debug
	for (int i=0; i<thread_tnt_map_size; i++){
		cout << c_c[thread_tnt_map[i]] << endl;
	}*/

    // Copy vectors from host to device global memory
    hipMemcpy(d_thread_tnt_map, thread_tnt_map, thread_tnt_map_size*sizeof(int), hipMemcpyHostToDevice);

    // Step2: find all final_results associated with each c_c
    tnt_results <<< num_blocks_per_grid, num_threads_per_block 
    			>>> (dcc, dcn, dno, c_c_size, c_n_size, n_o_size, d_thread_results, 
					 d_thread_tnt_map, thread_tnt_map_size, d_final_results, final_result_size, gridSize);

    // Set final_results
   	// Copy vectors from device to host
    hipMemcpy(final_results, d_final_results, final_result_size*15*sizeof(int), hipMemcpyDeviceToHost);

	// Free host memory
	free(thread_results);

    // Free device memory
    hipFree(dcc); 
    hipFree(dcn);
    hipFree(dno); 
    hipFree(d_thread_results);
    hipFree(d_final_results);
    hipFree(d_thread_tnt_map);
}